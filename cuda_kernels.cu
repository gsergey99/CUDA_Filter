#include "hip/hip_runtime.h"
/*********************************************
*   Project: Práctica de Computadores Avanzados 
*
*   Program name: cuda_kernels.cu
*
*   Author: Sergio Jiménez
*
*   Date created: 11-12-2020
*
*   Porpuse: Gestión para la realización de varios filtros y comparativas entre CUDA y OpenCV
*
*   Revision History: Reflejado en el repositorio de GitHub
|*********************************************/

#include <thread>
#include <chrono>
#include <string.h>
#include <ctime>
#include <iostream>
#include <math.h>
#include <typeinfo>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/stitching.hpp>
#include <opencv2/core.hpp>
#include "include/colours.h"

#define N 25.0 // Number of threads
#define DIM_KERNEL 3 // Dimension of Sobel kernel

/*Global variables for the  sobel kernel's gradient*/
__device__ const int kernel_sobel_x[DIM_KERNEL][DIM_KERNEL]={{-1,0,1},{-2,0,2},{-1,0,1}};
__device__ const int kernel_sobel_y[DIM_KERNEL][DIM_KERNEL]={{-1,-2,-1},{0,0,0},{1,2,1}};

__device__ const int kernel_sharpen[DIM_KERNEL][DIM_KERNEL] = {{0,-1,0},{-1,5,-1},{0,-1,0}};


void sobelFilterCPU(cv::Mat src_image, cv::Mat dest_image, const int width, const int height){
  for(int x = 1; x < src_image.rows-1; x++) {
      for(int y = 1; y < src_image.cols-1; y++) {

        float dy = (-1*src_image.data[(x-1)*height + (y-1)]) + (-2*src_image.data[x*height+(y-1)]) + (-1*src_image.data[(x+1)*height+(y-1)]) +
            (src_image.data[(x-1)*height + (y+1)]) + (2*src_image.data[x*height+(y+1)]) + (src_image.data[(x+1)*height+(y+1)]);
            
        float dx = (src_image.data[(x-1)*height + (y-1)]) + (2*src_image.data[(x-1)*height+y]) + (src_image.data[(x-1)*height+(y+1)]) +
        (-1*src_image.data[(x+1)*height + (y-1)]) + (-2*src_image.data[(x+1)*height+y]) + (-1*src_image.data[(x+1)*height+(y+1)]);
          
          dest_image.at<uchar>(x,y) = sqrt( (dx*dx) + (dy*dy) ) > 255 ? 255 : sqrt( (dx*dx) + (dy*dy) );
      }
  }
}

void sharpenFilterCPU(cv::Mat src_image, cv::Mat dest_image, const int width, const int height){
  float result;
  for(int x = 1; x < src_image.rows-1; x++) {
      for(int y = 1; y < src_image.cols-1; y++) {

        result = (-1 * src_image.data[(x-1)*height + y]) + (-1 * src_image.data[x*height+(y-1)]) + (5 * src_image.data[x*height+y]) 
        + (-1 * src_image.data[x*height+(y+1)]) + (-1 *src_image.data[(x+1)*height + y]);
        
          
          dest_image.at<uchar>(x,y) = (char)result;
      }
  }
}


__global__ void sobelKernelCUDA(unsigned char* src_image, unsigned char* dest_image, int width, int height){
     
    float dx,dy,result;

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    
    dx = (kernel_sobel_x[0][0] * src_image[(x-1)*height + (y-1)]) + (kernel_sobel_x[0][1] * src_image[(x-1)*height + y]) + (kernel_sobel_x[0][2] * src_image[(x-1)*height+(y+1)]) +
    (kernel_sobel_x[1][0] * src_image[x*height+(y-1)]) + (kernel_sobel_x[1][1] * src_image[x*height+y]) +(kernel_sobel_x[1][2] * src_image[x*height+(y+1)]) + 
    (kernel_sobel_x[2][0] * src_image[(x+1)*height +(y-1)]) + (kernel_sobel_x[2][1] *src_image[(x+1)*height + y]) + (kernel_sobel_x[2][2] * src_image[(x+1)*height + (y+1)]);
    
    dy = (kernel_sobel_y[0][0] * src_image[(x-1)*height + (y-1)]) + (kernel_sobel_y[0][1] * src_image[(x-1)*height + y]) + (kernel_sobel_y[0][2] * src_image[(x-1)*height+(y+1)]) +
    (kernel_sobel_y[1][0] * src_image[x*height+(y-1)]) + (kernel_sobel_y[1][1] * src_image[x*height+y]) +(kernel_sobel_y[1][2] * src_image[x*height+(y+1)]) + 
    (kernel_sobel_y[2][0] * src_image[(x+1)*height +(y-1)]) + (kernel_sobel_y[2][1] *src_image[(x+1)*height + y]) + (kernel_sobel_y[2][2] * src_image[(x+1)*height + (y+1)]);
    
    result = sqrt((pow(dx,2))+ (pow(dy,2)));

    /*Noise suppression*/
    if (result > 255) result = 255;
    if (result < 0) result = 0;

    dest_image[x*height+y] = result;

}

__global__ void sharpenKernelCUDA(unsigned char* src_image, unsigned char* dest_image, int width, int height){
   
  float result;

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  
  result = (kernel_sharpen[0][0] * src_image[(x-1)*height + (y-1)]) + (kernel_sharpen[0][1] * src_image[(x-1)*height + y]) + (kernel_sharpen[0][2] * src_image[(x-1)*height+(y+1)]) +
  (kernel_sharpen[1][0] * src_image[x*height+(y-1)]) + (kernel_sharpen[1][1] * src_image[x*height+y]) +(kernel_sharpen[1][2] * src_image[x*height+(y+1)]) + 
  (kernel_sharpen[2][0] * src_image[(x+1)*height +(y-1)]) + (kernel_sharpen[2][1] *src_image[(x+1)*height + y]) + (kernel_sharpen[2][2] * src_image[(x+1)*height + (y+1)]);
  
  /*Noise suppression*/
  if (result > 255) result = 255;
  if (result < 0) result = 0;

  dest_image[x*height+y] = result;
}


hipError_t testCuErr(hipError_t result){
  if (result != hipSuccess) {
    printf(FCYN("[CUDA MANAGER] CUDA Runtime Error: %s\n"), hipGetErrorString(result));
    assert(result == hipSuccess);      
  }
  return result;
}

int main(int argc,char * argv[]){
    
    int rows, cols;
    cv::Mat src_image, dest_image_cpu;
            
    if (argc!=3){
        std::cout << FRED("[MANAGER] The number of arguments are incorrect, please insert <image> <filter name: sobel, sharpen>") << std::endl;
        return 1;
    }

    if (strcmp(argv[2], "") == 0){
      std::cout << FRED("[MANAGER] The <filter name: sobel, sharpen> is not indicated") << std::endl;
      return 1;
    }
    
    src_image = cv::imread(argv[1]);
    
    if (src_image.empty()){
        std::cout << FRED("[MANAGER] There is a problem reading the image ")<< src_image << std::endl;
        return 1;
    }

    cv::cvtColor(src_image, src_image, cv::COLOR_RGB2GRAY);
    dest_image_cpu = cv::Mat::zeros(src_image.size(),src_image.type());
    
    cols = src_image.cols;
    rows = src_image.rows;

    std::cout << FCYN("[MANAGER] Using Image ") << argv[1] << FCYN(" | ROWS = ") <<  rows << FCYN(" COLS = ") << cols << std::endl;
    
    auto start_time = std::chrono::system_clock::now();
    if (strcmp(argv[2], "sobel") == 0)   sobelFilterCPU(src_image, dest_image_cpu, rows, cols);
    if (strcmp(argv[2], "sharpen") == 0)   sharpenFilterCPU(src_image, dest_image_cpu, rows, cols);

    std::chrono::duration<double> time_cpu = std::chrono::system_clock::now() - start_time;
    
    /*CUDA PART*/
    
    unsigned char *d_image, *h_image;
    int size = rows * cols;
    
    testCuErr(hipMalloc((void**)&d_image, size));
    testCuErr(hipMalloc((void**)&h_image, size));

    testCuErr(hipMemcpy(d_image,src_image.data,size, hipMemcpyHostToDevice));
    hipMemset(h_image, 0, size);
    
    dim3 threadsPerBlock(N, N, 1);
    dim3 numBlocks((int)ceil(rows/N), (int)ceil(cols/N), 1);

    start_time = std::chrono::system_clock::now();
    if (strcmp(argv[2], "sobel") == 0)  sobelKernelCUDA <<<numBlocks, threadsPerBlock>>> (d_image, h_image, rows, cols);
    if (strcmp(argv[2], "sharpen") == 0) sharpenKernelCUDA <<<numBlocks, threadsPerBlock>>> (d_image, h_image, rows, cols);

      testCuErr(hipGetLastError());
    std::chrono::duration<double> time_gpu = std::chrono::system_clock::now() - start_time;

    hipMemcpy(src_image.data, h_image, size, hipMemcpyDeviceToHost);
    hipFree(d_image); 
    hipFree(h_image);
    
    cv::imshow("CPU Filter", dest_image_cpu);
    cv::imshow("CUDA Filter",src_image);

    std::cout << FYEL("[MANAGER] Time CPU ") << time_cpu.count() * 1000 << FYEL(" milliseconds ") << std::endl;
    std::cout << FYEL("[MANAGER] Time GPU ") << time_gpu.count() * 1000 << FYEL(" milliseconds ") << std::endl;

    int k = cv::waitKey(0); // Wait for a keystroke in the windows

  return 0;
  }
