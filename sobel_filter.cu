#include "hip/hip_runtime.h"
/*********************************************
*   Project: Práctica de Computadores Avanzados 
*
*   Program name: sobel_filter.cu
*
*   Author: Sergio Jiménez
*
*   Date created: 11-12-2020
*
*   Porpuse: Gestión para la realización de un filtro Sobel
*
*   Revision History: Reflejado en el repositorio de GitHub
|*********************************************/

#include <thread>
#include <chrono>
#include <time.h>
#include <iostream>
#include <math.h>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/stitching.hpp>
#include <opencv2/core.hpp>

#define N 100000 // Number of threads
#define tb 512  // Block size
#define DIM_KERNEL 3 // Dimention of the kernel

__global__ void sobelKernel(unsigned char * src_image, int width, int height){
    const int kernel_x[DIM_KERNEL][DIM_KERNEL]={{-1,0,1},{-2,0,2},{-1,0,1}};
    const int kernel_y[DIM_KERNEL][DIM_KERNEL]={{-1,-2,-1},{0,0,0},{1,2,1}};
    int x,y,dx,dy,i,j, result;
    dx = 0;
    dy = 0;
    
    //Pensar en un código de manera secuencial
    result = sqrt((pow(dx,2))+ (pow(dy,2)));


    }




hipError_t testCuErr(hipError_t result){
  if (result != hipSuccess) {
    printf("CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);      // si no se cumple, se aborta el programa
  }
  return result;
}


int main(int argc,char * argv[]){
    
    int rows, cols;
    
    if (argc!=2){
        std::cout << "[MANAGER] The number of arguments are incorrect, please insert <image>" << std::endl;
        return 1;
    }

    cv::Mat src_image = cv::imread(argv[1]);
    if (src_image.empty()){
        std::cout << "[MANAGER] There is a problem reading the image "<< src_image << std::endl;
        return 1;
    }

    cv::cvtColor(src_image, src_image, cv::COLOR_RGB2GRAY);
    
    cols = src_image.cols;
    rows = src_image.rows;

    std::cout << "[MANAGER] Using Image " << argv[1] << " | ROWS = " <<  rows << " COLS = " << cols << std::endl;
    
    unsigned char *d_image, *h_image;
    int size = rows * cols * sizeof(unsigned char);
    h_image = (unsigned char*) malloc(size);

    testCuErr(hipMalloc((void**)&d_image,size));
    testCuErr(hipMemcpy(d_image,src_image.data,rows*cols*sizeof(unsigned char), hipMemcpyHostToDevice));

    int dg = (N+tb-1)/tb; if (dg>65535) dg=65535;



    //cv::imshow("Sobel Filter", src_image);





    //int k = cv::waitKey(0); // Wait for a keystroke in the window

    return 0;

}